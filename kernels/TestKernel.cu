#include "hip/hip_runtime.h"
__kernel void sampleKernel(
    const int2 imageSize,
    __global const int *img,
    __global const int *filter,
    __global int *result)
{
    int FILTER_SIZE = 3;
    int x = get_global_id(0);
    int y = get_global_id(1);

    int width = imageSize.x;
    int height = imageSize.y;

    int sum = 0;
    for(int filterY=0; filterY < FILTER_SIZE; filterY++) {
        for(int filterX=0; filterX < FILTER_SIZE; filterX++) {
            sum += img[ mul24((y - 1 + filterY), width) + x - 1 + filterX ] * filter[ (filterY * FILTER_SIZE) + filterX ];
        }
    }

    if(y + 1 < height && x + 1 < width) {
        result[mul24((y), width) + x] = sum / 16;
    }
}